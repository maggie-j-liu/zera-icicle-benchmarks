#include "hip/hip_runtime.h"

// clang++ -O2 -o vector_add_icicle vector_add_icicle.cu -Iinclude -I/home/magpie/icicle-install/icicle/include -I/usr/local/cuda-12.9/include -L/usr/local/cuda-12.9/lib64 -lcudart -lcuda --cuda-path=/usr/local/cuda-12.9 --cuda-gpu-arch=sm_90 -L/home/magpie/icicle-install/icicle/lib -licicle_device -licicle_field_bls12_381 -licicle_curve_bls12_381 -Wl,-rpath,/home/magpie/icicle-install/icicle/lib/
#include <iostream>

#include "icicle/runtime.h"
#include "icicle/vec_ops.h"
#include "icicle/curves/params/bls12_381.h"

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

using namespace bls12_381; 

__global__ void vector_add_kernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = a[idx] + b[idx]; // scalar_t defines operator+
    }
}

void cuda_vector_add(const scalar_t* h_a, const scalar_t* h_b, scalar_t* h_out, size_t n) {
    scalar_t *d_a, *d_b, *d_out;

    // Allocate device memory
    hipMalloc(&d_a, n * sizeof(scalar_t));
    hipMalloc(&d_b, n * sizeof(scalar_t));
    hipMalloc(&d_out, n * sizeof(scalar_t));

    // Copy inputs
    hipMemcpy(d_a, h_a, n * sizeof(scalar_t), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * sizeof(scalar_t), hipMemcpyHostToDevice);

    // Kernel launch config
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    hipProfilerStart();

    // Launch kernel
    vector_add_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_out, n);

    hipProfilerStop();

    // Sync and copy result back
    hipDeviceSynchronize();
    hipMemcpy(h_out, d_out, n * sizeof(scalar_t), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
}

int main() {
    icicle_load_backend_from_env_or_default();

    Device device_cpu = {"CPU", 0};
    Device device_gpu = {"CUDA", 0};
    if (icicle_is_device_available("CUDA") != eIcicleError::SUCCESS) {
      std::cout << "CUDA not available, using CPU only" << std::endl;
      device_gpu = device_cpu;
    }
    
    size_t n = 1 << 23; // ~1 million elements
    auto h_a = std::make_unique<scalar_t[]>(n);
    auto h_b = std::make_unique<scalar_t[]>(n);
    auto h_out = std::make_unique<scalar_t[]>(n);

    // Fill with random BN254 scalars
    scalar_t::rand_host_many(h_a.get(), n);
    scalar_t::rand_host_many(h_b.get(), n);

    // Run kernel
    cuda_vector_add(h_a.get(), h_b.get(), h_out.get(), n);

    return 0;
}